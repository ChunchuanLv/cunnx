#include "hip/hip_runtime.h"
#include "utils.h"



__global__ void mv(
float *m,
float *v,
float *o,
int size0,int size1) {

    
};

__global__ void CudaQUICK_PRODUCT_kernel(
THCState *state,
float *input1,
float *input2,
float *output,
int size0,int size1,int c = 2) {
if (c==0) return;
    c--;
const  float alpha = 1.0;
const  float beta = 0.0;
hipblasSgemv(state->currentBlasHandle, HIPBLAS_OP_N, size0, size1,   & alpha,input1+threadIdx.x, 1,input2+blockIdx.x*blockDim.x+threadIdx.x, 1,& beta, output+blockIdx.x*blockDim.x+threadIdx.x,1);
 //   CudaQUICK_PRODUCT_kernel<<<size0, size1>>>(state,input1,input2,output,size0,size1,c)
    
};


static int  CudaQUICK_PRODUCT(lua_State *L)
{
THCudaTensor *input1 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
THCudaTensor *input2 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
int length1 = luaL_checkint(L, 4);
int length2 = luaL_checkint(L, 5);
int length3 = luaL_checkint(L, 6);
THCState *state = getCutorchState(L);

int size0 = input1->size[0];
int size1 = input1->size[1];
CudaQUICK_PRODUCT_kernel <<<length1, length2>>>(
state,
THCudaTensor_data(state,input1),
THCudaTensor_data(state,input2),
THCudaTensor_data(state,output),size0,size1
);
/*
const  float alpha = 1.0;
const  float beta = 0.0;
hipblasSgemv(state->currentBlasHandle, HIPBLAS_OP_N, size0, size1,   & alpha,THCudaTensor_data(state,input1), 1,THCudaTensor_data(state,input2), 1,& beta,THCudaTensor_data(state,output),1);
*/
THCudaCheck(hipGetLastError());
return 1;
}

static const struct luaL_Reg cunnx_CudaQuickProduct__ [] = {
{"CudaQuickProduct_updateOutput", CudaQUICK_PRODUCT},
{NULL, NULL}
};

static void cunnx_CudaQuickProduct_init(lua_State *L)
{
luaT_pushmetatable(L, "torch.CudaTensor");
luaT_registeratname(L, cunnx_CudaQuickProduct__, "nn");
lua_pop(L,1);
}
